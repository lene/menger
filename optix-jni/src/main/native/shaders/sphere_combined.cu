#include "hip/hip_runtime.h"
#include <optix.h>
#include "../include/OptiXData.h"

extern "C" {
    __constant__ Params params;
}

// Constants for ray tracing
namespace Constants {
    constexpr float MAX_RAY_DISTANCE = 1e16f;
    constexpr float COLOR_SCALE_FACTOR = 255.99f;  // Slightly less than 256 to avoid overflow
}

// Device-side vector math helper functions
__device__ inline float3 normalize(float3 v) {
    const float len = sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
    return make_float3(v.x / len, v.y / len, v.z / len);
}

__device__ inline float dot(float3 a, float3 b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ inline float3 operator+(float3 a, float3 b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ inline float3 operator*(float3 v, float s) {
    return make_float3(v.x * s, v.y * s, v.z * s);
}

//==============================================================================
// Ray generation shader - generates primary rays from camera
//==============================================================================
extern "C" __global__ void __raygen__rg() {
    // Get ray generation data (camera parameters)
    const RayGenData* raygen_data = reinterpret_cast<RayGenData*>(optixGetSbtDataPointer());

    // Get pixel coordinates
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    // Calculate normalized device coordinates [-1, 1]
    const float u = (static_cast<float>(idx.x) + 0.5f) / static_cast<float>(dim.x) * 2.0f - 1.0f;
    const float v = (static_cast<float>(idx.y) + 0.5f) / static_cast<float>(dim.y) * 2.0f - 1.0f;

    // Construct ray direction from camera basis vectors
    const float3 ray_origin = make_float3(
        raygen_data->cam_eye[0],
        raygen_data->cam_eye[1],
        raygen_data->cam_eye[2]
    );

    const float3 camera_u = make_float3(raygen_data->camera_u[0], raygen_data->camera_u[1], raygen_data->camera_u[2]);
    const float3 camera_v = make_float3(raygen_data->camera_v[0], raygen_data->camera_v[1], raygen_data->camera_v[2]);
    const float3 camera_w = make_float3(raygen_data->camera_w[0], raygen_data->camera_w[1], raygen_data->camera_w[2]);

    const float3 ray_direction = normalize(camera_u * u + camera_v * v + camera_w);

    // Trace ray
    unsigned int p0, p1, p2;  // Payload for RGB color
    optixTrace(
        params.handle,                     // Acceleration structure
        ray_origin,                        // Ray origin
        ray_direction,                     // Ray direction
        0.0f,                              // tmin
        Constants::MAX_RAY_DISTANCE,       // tmax
        0.0f,                              // rayTime
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_NONE,
        0,                       // SBT offset
        1,                       // SBT stride
        0,                       // missSBTIndex
        p0, p1, p2               // Payload
    );

    // Convert payload to RGBA
    const unsigned int r = p0;
    const unsigned int g = p1;
    const unsigned int b = p2;

    // Write to output buffer
    const unsigned int pixel_index = idx.y * params.image_width + idx.x;
    params.image[pixel_index * 4 + 0] = static_cast<unsigned char>(r);
    params.image[pixel_index * 4 + 1] = static_cast<unsigned char>(g);
    params.image[pixel_index * 4 + 2] = static_cast<unsigned char>(b);
    params.image[pixel_index * 4 + 3] = 255;  // Alpha
}

//==============================================================================
// Miss shader - returns background color when ray hits nothing
//==============================================================================
extern "C" __global__ void __miss__ms() {
    // Get miss data (background color)
    const MissData* miss_data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());

    // Convert float [0,1] to unsigned int [0,255]
    const unsigned int r = static_cast<unsigned int>(miss_data->r * Constants::COLOR_SCALE_FACTOR);
    const unsigned int g = static_cast<unsigned int>(miss_data->g * Constants::COLOR_SCALE_FACTOR);
    const unsigned int b = static_cast<unsigned int>(miss_data->b * Constants::COLOR_SCALE_FACTOR);

    // Set payload
    optixSetPayload_0(r);
    optixSetPayload_1(g);
    optixSetPayload_2(b);
}

//==============================================================================
// Closest hit shader - computes Lambertian shading for sphere
//==============================================================================
extern "C" __global__ void __closesthit__ch() {
    // Get hit group data (light parameters)
    const HitGroupData* hit_data = reinterpret_cast<HitGroupData*>(optixGetSbtDataPointer());

    // Get hit point (ray origin + t * ray direction)
    const float t = optixGetRayTmax();
    const float3 ray_origin = optixGetWorldRayOrigin();
    const float3 ray_direction = optixGetWorldRayDirection();

    const float3 hit_point = ray_origin + ray_direction * t;

    // LIMITATION: Currently assumes sphere is centered at origin (0,0,0)
    // For a sphere at the origin, the surface normal at any point is simply
    // the normalized position vector from the origin to that point.
    //
    // Future Enhancement: To support arbitrary sphere centers, we would need to:
    // 1. Add sphere center position to HitGroupData
    // 2. Calculate normal as: normalize(hit_point - sphere_center)
    //
    // This limitation is acceptable for Phase 2 which focuses on basic OptiX
    // integration. Phase 3+ can extend this to handle multiple spheres at
    // arbitrary positions.
    const float3 normal = normalize(hit_point);

    // Light direction (negated for dot product)
    const float3 light_dir = make_float3(
        -hit_data->light_dir[0],
        -hit_data->light_dir[1],
        -hit_data->light_dir[2]
    );

    // Lambertian shading: max(0, N · L)
    const float ndotl = fmaxf(0.0f, dot(normal, light_dir));

    // Apply light intensity and material color (white)
    const float intensity = ndotl * hit_data->light_intensity;

    // Convert to RGB [0, 255]
    const unsigned int color = static_cast<unsigned int>(intensity * Constants::COLOR_SCALE_FACTOR);

    // Set payload (grayscale for now)
    optixSetPayload_0(color);
    optixSetPayload_1(color);
    optixSetPayload_2(color);
}
