#include "hip/hip_runtime.h"
#include <optix.h>
#include "../include/OptiXData.h"

// Closest hit shader - computes Lambertian shading for sphere
extern "C" __global__ void __closesthit__ch() {
    // Get hit group data (light parameters)
    const HitGroupData* hit_data = reinterpret_cast<HitGroupData*>(optixGetSbtDataPointer());

    // Get geometric normal (for built-in sphere primitive, it's provided by OptiX)
    const float3 world_normal = optixGetWorldRayDirection();  // Placeholder - will be computed from hit point

    // Get hit point (ray origin + t * ray direction)
    const float t = optixGetRayTmax();
    const float3 ray_origin = optixGetWorldRayOrigin();
    const float3 ray_direction = optixGetWorldRayDirection();

    const float3 hit_point = make_float3(
        ray_origin.x + t * ray_direction.x,
        ray_origin.y + t * ray_direction.y,
        ray_origin.z + t * ray_direction.z
    );

    // For analytic sphere intersection, we need sphere center from SBT or compute normal
    // For now, assume sphere at origin - normal is just hit_point normalized
    float3 normal = hit_point;
    const float len = sqrtf(normal.x * normal.x + normal.y * normal.y + normal.z * normal.z);
    normal.x /= len;
    normal.y /= len;
    normal.z /= len;

    // Light direction (negated for dot product)
    const float3 light_dir = make_float3(
        -hit_data->light_dir[0],
        -hit_data->light_dir[1],
        -hit_data->light_dir[2]
    );

    // Lambertian shading: max(0, N · L)
    float ndotl = normal.x * light_dir.x + normal.y * light_dir.y + normal.z * light_dir.z;
    ndotl = fmaxf(0.0f, ndotl);

    // Apply light intensity and material color (white)
    const float intensity = ndotl * hit_data->light_intensity;

    // Convert to RGB [0, 255]
    const unsigned int color = static_cast<unsigned int>(intensity * 255.99f);

    // Set payload (grayscale for now)
    optixSetPayload_0(color);
    optixSetPayload_1(color);
    optixSetPayload_2(color);
}
